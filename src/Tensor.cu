#include "hip/hip_runtime.h"
#include "NeuZephyr/Tensor.cuh"
#include "NeuZephyr/utils.cuh"
#include "NeuZephyr/OperationKernels.cuh"
#include "NeuZephyr/NeuZephyrCudaErrorHandling.cuh"
#include <hiprand.h>

namespace nz::data {
    /**
     * @brief Overloads the `<<` operator to print the tensor's data to an output stream.
     *
     * This function is a friend of the `Tensor` class and provides an overloaded version of
     * the output stream operator (`<<`) to print the contents of a tensor to the specified
     * output stream (e.g., `std::cout` or a file stream).
     *
     * The tensor's data is first copied from GPU memory to host memory for printing, and then the data
     * is printed in a 2D matrix format. Each row of the tensor is printed on a new line, and each element
     * in a row is separated by a space. Each row is enclosed in square brackets.
     *
     * @param os The output stream to which the tensor will be printed.
     * @param tensor The tensor whose contents will be printed.
     * @return The output stream (`os`) after the tensor has been printed, allowing for chaining of operations.
     *
     * @note
     * - This operator works by accessing the tensor's private data members (e.g., `_data`) directly.
     * - The tensor's data is assumed to be in a valid state (i.e., properly allocated in GPU memory) before printing.
     * - The function copies the tensor's data from device (GPU) memory to host (CPU) memory using `hipMemcpy`, which
     *   may introduce performance overhead for large tensors.
     *
     * @code
     * ```cpp
     * Tensor tensor({2, 3});
     * tensor.fill(1.0f);  // Fill the tensor with 1.0f
     * std::cout << tensor << std::endl;  // Prints the tensor to standard output in matrix format
     * ```
     * @endcode
     */
    std::ostream& operator<<(std::ostream& os, const Tensor& tensor) {
        auto* data = static_cast<Tensor::value_type*>(malloc(tensor._size * sizeof(Tensor::value_type)));
        CHECK(hipMemcpy(data, tensor._data, tensor._size * sizeof(Tensor::value_type), hipMemcpyDeviceToHost));
        std::ostream_iterator<Tensor::value_type> output_iterator(os, " ");
        for (int i = 0; i < tensor._shape[0]; ++i) {
            const auto it = data + i * tensor._shape[1];
            const auto it_end = it + tensor._shape[1];
            os << "[";
            std::copy(it, it_end, output_iterator);
            os << "]";
            os << std::endl;
        }
        if (tensor._requires_grad) {
            os << "Gradient: " << std::endl;
            auto* grad = static_cast<Tensor::value_type*>(malloc(tensor._size * sizeof(Tensor::value_type)));
            CHECK(hipMemcpy(grad, tensor._grad, tensor._size * sizeof(Tensor::value_type), hipMemcpyDeviceToHost));
            for (int i = 0; i < tensor._shape[0]; ++i) {
                const auto it = grad + i * tensor._shape[1];
                const auto it_end = it + tensor._shape[1];
                os << "[";
                std::copy(it, it_end, output_iterator);
                os << "]";
                os << std::endl;
            }
            free(grad);
        }
        free(data);
        return os;
    }

    /**
     * @brief Overloads the `>>` operator to read a tensor's data from an input stream.
     *
     * This function is a friend of the `Tensor` class and provides an overloaded version of
     * the input stream operator (`>>`) to read the contents of a tensor from the specified
     * input stream (e.g., `std::cin` or a file stream).
     *
     * The function reads the tensor's data element by element from the input stream and stores
     * the values in a temporary buffer. Once all the data has been read, it is copied from the
     * host memory back into the tensor's GPU memory using `hipMemcpy`.
     *
     * @param is The input stream from which the tensor's data will be read.
     * @param tensor The tensor to which the data will be read.
     * @return The input stream (`is`) after reading the tensor's data, allowing for chaining of operations.
     *
     * @note
     * - This operator works by reading data from the input stream and storing it in a temporary buffer on the host.
     * - The function assumes that the input data matches the size of the tensor. If the data is malformed or does not
     *   match, the behavior may be undefined.
     * - After reading, the data is copied from host memory back into the tensor's GPU memory.
     *
     * @code
     * ```cpp
     * Tensor tensor({2, 3});
     * std::cin >> tensor;  // Reads the tensor's data from standard input
     * ```
     * @endcode
     */
    std::istream& operator>>(std::istream& is, const Tensor& tensor) {
        auto* data = static_cast<Tensor::value_type*>(malloc(tensor._size * sizeof(Tensor::value_type)));
        for (int i = 0; i < tensor._size; ++i) {
            is >> data[i];
        }
        CHECK(hipMemcpy(tensor._data, data, tensor._size * sizeof(Tensor::value_type), hipMemcpyHostToDevice));
        free(data);
        return is;
    }

    /**
     * @brief Multiplies a tensor by a scalar (element-wise multiplication).
     *
     * This function is a friend of the `Tensor` class and provides an overloaded version of
     * the multiplication operator (`*`) to multiply each element of the tensor by a scalar value.
     * It performs element-wise multiplication, where every element in the tensor is multiplied
     * by the given scalar.
     *
     * @param lhs The scalar value to multiply each element of the tensor by.
     * @param rhs The tensor whose elements will be multiplied by the scalar.
     * @return A new tensor containing the result of the element-wise multiplication.
     *
     * This function uses a CUDA kernel (`ScalarMul`) to perform the element-wise multiplication in parallel
     * on the GPU. The result is stored in a new tensor, which is returned.
     *
     * @note
     * - This operator does not modify the original tensor. Instead, it returns a new tensor that contains the
     *   result of the element-wise multiplication.
     * - The function assumes that the tensor is already in a valid state and that the tensor's data is in GPU memory.
     *
     * @code
     * ```cpp
     * Tensor tensor({2, 3});
     * tensor.fill(1.0f);  // Fill the tensor with 1.0f
     * float scalar = 2.0f;
     * Tensor result = scalar * tensor;  // Multiply each element of the tensor by 2.0f
     * std::cout << result << std::endl;  // Print the resulting tensor
     * ```
     * @endcode
     */
    Tensor operator*(const Tensor::value_type lhs, const Tensor& rhs) {
        Tensor result(rhs._shape, rhs._requires_grad);
        dim3 block(256);
        dim3 grid((rhs._size + block.x - 1) / block.x);
        krnl::ScalarMul(grid, block, result._data, rhs._data, lhs, rhs._size);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    /**
     * @brief Multiplies a tensor by a scalar (element-wise multiplication).
     *
     * This function is a friend of the `Tensor` class and provides an overloaded version of
     * the multiplication operator (`*`) to multiply each element of the tensor by a scalar value.
     * It performs element-wise multiplication, where every element in the tensor is multiplied
     * by the given scalar.
     *
     * @param lhs The tensor whose elements will be multiplied by the scalar.
     * @param rhs The scalar value to multiply each element of the tensor by.
     * @return A new tensor containing the result of the element-wise multiplication.
     *
     * This function uses a CUDA kernel (`ScalarMul`) to perform the element-wise multiplication in parallel
     * on the GPU. The result is stored in a new tensor, which is returned.
     *
     * @note
     * - This operator does not modify the original tensor. Instead, it returns a new tensor that contains the
     *   result of the element-wise multiplication.
     * - The function assumes that the tensor is already in a valid state and that the tensor's data is in GPU memory.
     *
     * @code
     * ```cpp
     * Tensor tensor({2, 3});
     * tensor.fill(1.0f);  // Fill the tensor with 1.0f
     * float scalar = 2.0f;
     * Tensor result = tensor * scalar;  // Multiply each element of the tensor by 2.0f
     * std::cout << result << std::endl;  // Print the resulting tensor
     * ```
     * @endcode
     */
    Tensor operator*(const Tensor& lhs, const Tensor::value_type rhs) {
        Tensor result(lhs._shape, lhs._requires_grad);
        dim3 block(256);
        dim3 grid((lhs._size + block.x - 1) / block.x);
        krnl::ScalarMul(grid, block, result._data, lhs._data, rhs, lhs._size);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    /**
     * @brief Divides a tensor by a scalar (element-wise division).
     *
     * This function is a friend of the `Tensor` class and provides an overloaded version of
     * the division operator (`/`) to divide each element of the tensor by a scalar value.
     * It performs element-wise division, where every element in the tensor is divided by the given scalar.
     *
     * @param lhs The tensor whose elements will be divided by the scalar.
     * @param rhs The scalar value by which each element of the tensor will be divided.
     * @return A new tensor containing the result of the element-wise division.
     *
     * This function uses a CUDA kernel (`ScalarDiv`) to perform the element-wise division in parallel
     * on the GPU. The result is stored in a new tensor, which is returned.
     *
     * @note
     * - This operator does not modify the original tensor. Instead, it returns a new tensor that contains the
     *   result of the element-wise division.
     * - The function assumes that the tensor is already in a valid state and that the tensor's data is in GPU memory.
     * - Division by zero should be handled appropriately, and input tensors should be checked to ensure no element is zero.
     *
     * @code
     * ```cpp
     * Tensor tensor({2, 3});
     * tensor.fill(10.0f);  // Fill the tensor with 10.0f
     * float scalar = 2.0f;
     * Tensor result = tensor / scalar;  // Divide each element of the tensor by 2.0f
     * std::cout << result << std::endl;  // Print the resulting tensor
     * ```
     * @endcode
     */
    Tensor operator/(const Tensor& lhs, const Tensor::value_type rhs) {
        Tensor result(lhs._shape, lhs._requires_grad);
        dim3 block(256);
        dim3 grid((lhs._size + block.x - 1) / block.x);
        krnl::ScalarDiv(grid, block, result._data, lhs._data, rhs, lhs._size);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    /**
     * @brief Adds a scalar to a tensor (element-wise addition).
     *
     * This function is a friend of the `Tensor` class and provides an overloaded version of
     * the addition operator (`+`) to add a scalar value to each element of the tensor.
     * It performs element-wise addition, where every element in the tensor is increased by the given scalar.
     *
     * @param lhs The tensor whose elements will be added by the scalar.
     * @param rhs The scalar value to add to each element of the tensor.
     * @return A new tensor containing the result of the element-wise addition.
     *
     * This function uses a CUDA kernel (`ScalarAdd`) to perform the element-wise addition in parallel
     * on the GPU. The result is stored in a new tensor, which is returned.
     *
     * @note
     * - This operator does not modify the original tensor. Instead, it returns a new tensor that contains the
     *   result of the element-wise addition.
     * - The function assumes that the tensor is already in a valid state and that the tensor's data is in GPU memory.
     *
     * @code
     * ```cpp
     * Tensor tensor({2, 3});
     * tensor.fill(1.0f);  // Fill the tensor with 1.0f
     * float scalar = 2.0f;
     * Tensor result = tensor + scalar;  // Add 2.0f to each element of the tensor
     * std::cout << result << std::endl;  // Print the resulting tensor
     * ```
     * @endcode
     */
    Tensor operator+(const Tensor& lhs, const Tensor::value_type rhs) {
        Tensor result(lhs._shape, lhs._requires_grad);
        dim3 block(256);
        dim3 grid((lhs._size + block.x - 1) / block.x);
        krnl::ScalarAdd(grid, block, result._data, lhs._data, rhs, lhs._size);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    /**
     * @brief Adds a scalar to a tensor (element-wise addition).
     *
     * This function is a friend of the `Tensor` class and provides an overloaded version of
     * the addition operator (`+`) to add a scalar value to each element of the tensor.
     * It performs element-wise addition, where every element in the tensor is increased by the given scalar.
     *
     * @param lhs The scalar value to add to each element of the tensor.
     * @param rhs The tensor whose elements will be added by the scalar.
     * @return A new tensor containing the result of the element-wise addition.
     *
     * This function uses a CUDA kernel (`ScalarAdd`) to perform the element-wise addition in parallel
     * on the GPU. The result is stored in a new tensor, which is returned.
     *
     * @note
     * - This operator does not modify the original tensor. Instead, it returns a new tensor that contains the
     *   result of the element-wise addition.
     * - The function assumes that the tensor is already in a valid state and that the tensor's data is in GPU memory.
     *
     * @code
     * ```cpp
     * Tensor tensor({2, 3});
     * tensor.fill(1.0f);  // Fill the tensor with 1.0f
     * float scalar = 2.0f;
     * Tensor result = scalar + tensor;  // Add 2.0f to each element of the tensor
     * std::cout << result << std::endl;  // Print the resulting tensor
     * ```
     * @endcode
     */
    Tensor operator+(const Tensor::value_type lhs, const Tensor& rhs) {
        Tensor result(rhs._shape, rhs._requires_grad);
        dim3 block(256);
        dim3 grid((rhs._size + block.x - 1) / block.x);
        krnl::ScalarAdd(grid, block, result._data, rhs._data, lhs, rhs._size);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    /**
     * @brief Subtracts a scalar from a tensor (element-wise subtraction).
     *
     * This function is a friend of the `Tensor` class and provides an overloaded version of
     * the subtraction operator (`-`) to subtract a scalar value from each element of the tensor.
     * It performs element-wise subtraction, where every element in the tensor is decreased by the given scalar.
     *
     * @param lhs The tensor whose elements will have the scalar subtracted from them.
     * @param rhs The scalar value to subtract from each element of the tensor.
     * @return A new tensor containing the result of the element-wise subtraction.
     *
     * This function uses a CUDA kernel (`ScalarAdd`) to perform the element-wise subtraction in parallel
     * on the GPU. The result is stored in a new tensor, which is returned. The scalar is negated during the
     * operation to achieve subtraction.
     *
     * @note
     * - This operator does not modify the original tensor. Instead, it returns a new tensor that contains the
     *   result of the element-wise subtraction.
     * - The function assumes that the tensor is already in a valid state and that the tensor's data is in GPU memory.
     *
     * @code
     * ```cpp
     * Tensor tensor({2, 3});
     * tensor.fill(10.0f);  // Fill the tensor with 10.0f
     * float scalar = 2.0f;
     * Tensor result = tensor - scalar;  // Subtract 2.0f from each element of the tensor
     * std::cout << result << std::endl;  // Print the resulting tensor
     * ```
     * @endcode
     */
    Tensor operator-(const Tensor& lhs, const Tensor::value_type rhs) {
        Tensor result(lhs._shape, lhs._requires_grad);
        dim3 block(256);
        dim3 grid((lhs._size + block.x - 1) / block.x);
        krnl::ScalarAdd(grid, block, result._data, lhs._data, -rhs, lhs._size);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    /**
     * @brief Subtracts a tensor from a scalar (element-wise subtraction).
     *
     * This function is a friend of the `Tensor` class and provides an overloaded version of
     * the subtraction operator (`-`) to subtract each element of the tensor from a scalar value.
     * It performs element-wise subtraction, where every element in the tensor is subtracted from the given scalar.
     *
     * @param lhs The scalar value from which each element of the tensor will be subtracted.
     * @param rhs The tensor whose elements will be subtracted from the scalar.
     * @return A new tensor containing the result of the element-wise subtraction.
     *
     * This function uses a CUDA kernel (`ScalarAdd`) to perform the element-wise subtraction in parallel
     * on the GPU. The result is stored in a new tensor, which is returned. The scalar is negated during the
     * operation to achieve subtraction.
     *
     * @note
     * - This operator does not modify the original tensor. Instead, it returns a new tensor that contains the
     *   result of the element-wise subtraction.
     * - The function assumes that the tensor is already in a valid state and that the tensor's data is in GPU memory.
     * - The scalar is negated to perform subtraction, which results in `lhs - rhs` for each element in the tensor.
     *
     * @code
     * ```cpp
     * Tensor tensor({2, 3});
     * tensor.fill(10.0f);  // Fill the tensor with 10.0f
     * float scalar = 2.0f;
     * Tensor result = scalar - tensor;  // Subtract each element of the tensor from 2.0f
     * std::cout << result << std::endl;  // Print the resulting tensor
     * ```
     * @endcode
     */
    Tensor operator-(const Tensor::value_type lhs, const Tensor& rhs) {
        Tensor result(rhs._shape, rhs._requires_grad);
        dim3 block(256);
        dim3 grid((rhs._size + block.x - 1) / block.x);
        krnl::ScalarAdd(grid, block, result._data, rhs._data, -lhs, rhs._size);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    /**
     * @brief Applies the Softmax activation function to a tensor.
     *
     * This function is a friend of the `Tensor` class and applies the Softmax activation function element-wise
     * to the given tensor. The Softmax function converts the tensor into a probability distribution, where each
     * element is transformed into a value between 0 and 1, and the sum of all elements in the tensor equals 1.
     * The Softmax function is commonly used in the output layer of neural networks for multi-class classification tasks.
     *
     * The Softmax function for each element `x_i` in the tensor is computed as:
     *
     *     Softmax(x_i) = exp(x_i) / sum(exp(x_j) for all j)
     *
     * @param tensor The tensor to which the Softmax activation function will be applied.
     * @return The input tensor with the Softmax function applied element-wise.
     *
     * This function uses a CUDA kernel (`SummationExp`) to compute the sum of the exponentiated values in parallel
     * on the GPU, and then another kernel (`Softmax`) to apply the Softmax transformation. The result is stored in
     * the original tensor, which is returned.
     *
     * @note
     * - This operator modifies the original tensor by applying the Softmax transformation in-place.
     * - The function assumes that the tensor is already in a valid state and that the tensor's data is in GPU memory.
     * - The Softmax computation is performed in two stages: first by calculating the exponentiated values' sum,
     *   then applying the Softmax transformation.
     *
     * @code
     * ```cpp
     * Tensor tensor({2, 3});
     * tensor.fill(1.0f);  // Fill the tensor with values
     * Tensor result = Softmax(tensor);  // Apply the Softmax activation
     * std::cout << result << std::endl;  // Print the resulting tensor
     * ```
     * @endcode
     */
    Tensor Softmax(const Tensor& tensor) {
        const dim3 block(256);
        const dim3 grid((tensor._size + block.x - 1) / block.x);
        float* result_d;
        float* result_h;
        float sum = 0;
        hipMalloc(&result_d, grid.x * sizeof(Tensor::value_type));
        result_h = static_cast<float*>(malloc(grid.x * sizeof(Tensor::value_type)));
        krnl::SummationExp(grid, block, block.x / WARP_SIZE * sizeof(float), result_d, tensor._data, tensor._size);
        CHECK(hipMemcpy(result_h, result_d, grid.x * sizeof(Tensor::value_type), hipMemcpyDeviceToHost));
        for (int i = 0; i < grid.x; i++) {
            sum += result_h[i];
        }
        free(result_h);
        hipFree(result_d);
        krnl::Softmax(grid, block, tensor._data, tensor._data, sum, tensor._size);
        CHECK(hipDeviceSynchronize());
        return tensor;
    }


    // Constructors
    Tensor::Tensor() :
        _size(0), _shape({0, 0}), _data(nullptr), _grad(nullptr), _requires_grad(false) {
    }

    Tensor::Tensor(const shape_type& shape, const bool requires_grad) // NOLINT(*-pro-type-member-init)
        :
        _size(shape[0] * shape[1]), _shape(shape), _requires_grad(requires_grad) {
        CHECK(hipMalloc(&_data, _size * sizeof(value_type)));
        if (_requires_grad) {
            CHECK(hipMalloc(&_grad, _size * sizeof(value_type)));
        }
        else {
            _grad = nullptr;
        }
    }

    Tensor::Tensor(const shape_type& shape, const value_type* data, const bool requires_grad, const bool host) :
        _size(shape[0] * shape[1]), _shape(shape), _requires_grad(requires_grad) {
        CHECK(hipMalloc(&_data, _size * sizeof(value_type)));
        if (host) {
            CHECK(hipMemcpy(_data, data, _size * sizeof(value_type), hipMemcpyHostToDevice));
        }
        else {
            CHECK(hipMemcpy(_data, data, _size * sizeof(value_type), hipMemcpyDeviceToDevice));
        }
        if (_requires_grad) {
            CHECK(hipMalloc(&_grad, _size * sizeof(value_type)));
        }
        else {
            _grad = nullptr;
        }
    }

    Tensor::Tensor(const shape_type& shape, const std::initializer_list<value_type>& data, const bool requires_grad) :
        _size(shape[0] * shape[1]), _shape(shape), _requires_grad(requires_grad) {
        if (std::distance(data.begin(), data.end()) < _size) {
            throw std::invalid_argument("Initializer list size is less than the tensor size.");
        }
        CHECK(hipMalloc(&_data, _size * sizeof(value_type)));
        if (_requires_grad) {
            CHECK(hipMalloc(&_grad, _size * sizeof(value_type)));
        }
        else {
            _grad = nullptr;
        }
        auto host_buf = new value_type[_size];
        auto it = data.begin();
        for (auto i = 0; i < _size; ++i, ++it) {
            host_buf[i] = *it;
        }
        CHECK(hipMemcpy(_data, host_buf, _size * sizeof(value_type), hipMemcpyHostToDevice));
        delete[] host_buf;
    }

    // Copy and Move constructors
    Tensor::Tensor(const Tensor& other) :
        _size(other._size), _shape(other._shape), _requires_grad(other._requires_grad) {
        CHECK(hipMalloc(&_data, _size * sizeof(value_type)));
        CHECK(hipMemcpy(_data, other._data, _size * sizeof(value_type), hipMemcpyDeviceToDevice));
        if (_requires_grad) {
            CHECK(hipMalloc(&_grad, _size * sizeof(value_type)));
            CHECK(hipMemcpy(_grad, other._grad, _size * sizeof(value_type), hipMemcpyDeviceToDevice));
        }
        else {
            _grad = nullptr;
        }
    }

    Tensor::Tensor(Tensor&& other) noexcept(false):
        _size(other._size), _shape(std::move(other._shape)), _requires_grad(other._requires_grad) {
        CHECK(hipMalloc(&_data, _size * sizeof(value_type)));
        CHECK(hipMalloc(&_grad, _size * sizeof(value_type)));
        CHECK(hipMemcpy(_data, other._data, _size * sizeof(value_type), hipMemcpyDeviceToDevice));
        if (_requires_grad) {
            CHECK(hipMemcpy(_grad, other._grad, _size * sizeof(value_type), hipMemcpyDeviceToDevice));
        }
        other._data = nullptr;
        other._grad = nullptr;
    }

    Tensor& Tensor::operator=(const Tensor& other) {
        if (this != &other) {
            _size = other._size;
            _shape = other._shape;
            _requires_grad = other._requires_grad;
            CHECK(hipFree(_data));
            CHECK(hipMalloc((value_type**)&_data, _size * sizeof(value_type)));
            CHECK(hipMemcpy(_data, other._data, _size * sizeof(value_type), hipMemcpyDeviceToDevice));
            if (_requires_grad) {
                CHECK(hipFree(_grad));
                CHECK(hipMalloc((value_type**)&_grad, _size * sizeof(value_type)));
                CHECK(hipMemcpy(_grad, other._grad, _size * sizeof(value_type), hipMemcpyDeviceToDevice));
            }
        }
        return *this;
    }

    Tensor& Tensor::operator=(Tensor&& other) noexcept(false) {
        if (this != &other) {
            _size = other._size;
            _shape = std::move(other._shape);
            CHECK(hipMemcpy(_data, other._data, _size * sizeof(value_type), hipMemcpyDeviceToDevice));
            if (_requires_grad) {
                CHECK(hipMemcpy(_grad, other._grad, _size * sizeof(value_type), hipMemcpyDeviceToDevice));
            }
            other._data = nullptr;
            other._grad = nullptr;
        }
        return *this;
    }

    Tensor::~Tensor() noexcept(false) {
        CHECK(hipFree(_data));
        if (_requires_grad) {
            CHECK(hipFree(_grad));
        }
    }

    // Getter methods
    bool Tensor::requiresGrad() const noexcept { return _requires_grad; }
    Tensor::shape_type Tensor::shape() const noexcept { return _shape; }
    Tensor::size_type Tensor::size() const noexcept { return _size; }

    // Setter methods
    void Tensor::setRequiresGrad(const bool requires_grad) {
        if (requires_grad && _grad == nullptr) {
            CHECK(hipMalloc(reinterpret_cast<value_type**>(_grad), _size * sizeof(value_type)));
        }
        if (!requires_grad && _grad != nullptr) {
            CHECK(hipFree(_grad));
            _grad = nullptr;
        }
        _requires_grad = requires_grad;
    }

    void Tensor::dataInject(const std::initializer_list<value_type>& data, const bool grad) const {
        dataInject(data.begin(), data.end(), grad);
    }

    // Operations
    void Tensor::zeroGrad() const {
        if (_requires_grad) {
            CHECK(hipMemset(_grad, 0, _size * sizeof(value_type)));
        }
    }

    void Tensor::print() const {
        const std::ostream_iterator<value_type> output_iterator(std::cout, " ");
        auto* data = static_cast<value_type*>(malloc(_size * sizeof(value_type)));
        CHECK(hipMemcpy(data, _data, _size * sizeof(value_type), hipMemcpyDeviceToHost));
        for (size_type i = 0; i < _shape[0]; ++i) {
            const auto it = data + i * _shape[1];
            const auto end_it = it + _shape[1];
            std::cout << "[";
            std::copy(it, end_it, output_iterator);
            std::cout << "]";
            std::cout << std::endl;
        }
        free(data);
    }

    void Tensor::dataInject(const value_type* data, const bool grad) const {
        if (grad) {
            if (_requires_grad) {
                CHECK(hipMemcpy(_grad, data, _size * sizeof(value_type), hipMemcpyHostToDevice));
            }
            else {
                throw std::runtime_error("Tensor does not require gradients");
            }
        }
        else {
            CHECK(hipMemcpy(_data, data, _size * sizeof(value_type), hipMemcpyHostToDevice));
        }
    }

    void Tensor::randomize(unsigned long long seed) const {
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, seed);
        hiprandGenerateUniform(gen, _data, _size);
    }

    void Tensor::clear() const {
        CHECK(hipMemset(_data, 0, _size * sizeof(value_type)));
    }

    void Tensor::fill(const value_type value) const {
        auto* data = static_cast<value_type*>(malloc(_size * sizeof(value_type)));
        for (size_type i = 0; i < _size; ++i) {
            data[i] = value;
        }
        CHECK(hipMemcpy(_data, data, _size * sizeof(value_type), hipMemcpyHostToDevice));
        free(data);
    }

    void Tensor::fillGrad(const value_type value) const {
        auto* grad = static_cast<value_type*>(malloc(_size * sizeof(value_type)));
        for (size_type i = 0; i < _size; ++i) {
            grad[i] = value;
        }
        CHECK(hipMemcpy(_grad, grad, _size * sizeof(value_type), hipMemcpyHostToDevice));
        free(grad);
    }

    Tensor Tensor::operator+(const Tensor& other) const {
        if (_shape != other._shape) {
            throw std::invalid_argument("Tensor shapes do not match");
        }
        Tensor result(_shape, _requires_grad);
        const dim3 block(256);
        const dim3 grid((_size + block.x - 1) / block.x);
        krnl::MatrixAdd(grid, block, _data, other._data, result._data, _size);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    Tensor Tensor::operator-(const Tensor& other) const {
        if (_shape != other._shape) {
            throw std::invalid_argument("Tensor shapes do not match");
        }
        Tensor result(_shape, _requires_grad);
        const dim3 block(256);
        const dim3 grid((_size + block.x - 1) / block.x);
        krnl::MatrixSub(grid, block, _data, other._data, result._data, _size);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    Tensor Tensor::operator*(const Tensor& other) const {
        if (_shape[1] != other._shape[0]) {
            throw std::invalid_argument("Matrix shapes do not match");
        }
        Tensor result({_shape[0], other._shape[1]}, _requires_grad);
        const dim3 block(TILE_SIZE, TILE_SIZE);
        const dim3 grid((result._shape[1] + block.x - 1) / block.x, (result._shape[0] + block.y - 1) / block.y);
        krnl::GeneralMatrixMul(grid, block, _data, other._data, result._data, _shape[0], other._shape[1],
                               _shape[1]);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    void Tensor::reshape(const shape_type& shape) {
        if (shape[0] * shape[1] != _size) {
            WARN("Reshaping to a different size will cause data loss");
        }
        auto* temp = static_cast<value_type*>(malloc(_size * sizeof(value_type)));
        CHECK(hipMemcpy(temp, _data, _size * sizeof(value_type), hipMemcpyDeviceToHost));
        CHECK(hipFree(_data));

        value_type* temp_grad = nullptr;
        if (_requires_grad) {
            temp_grad = static_cast<value_type*>(malloc(_size * sizeof(value_type)));
            CHECK(hipMemcpy(temp_grad, _grad, _size * sizeof(value_type), hipMemcpyDeviceToHost));
            CHECK(hipFree(_grad));
        }

        const size_type size = _size;
        _size = shape[0] * shape[1];
        _shape = shape;

        CHECK(hipMalloc(&_data, _size * sizeof(value_type)));
        CHECK(hipMemset(_data, 0, _size * sizeof(value_type)));
        CHECK(hipMemcpy(_data, temp, size * sizeof(value_type), hipMemcpyHostToDevice));
        free(temp);

        if (_requires_grad) {
            CHECK(hipMalloc(&_grad, _size * sizeof(value_type)));
            CHECK(hipMemset(_grad, 0, _size * sizeof(value_type)));
            CHECK(hipMemcpy(_grad, temp_grad, size * sizeof(value_type), hipMemcpyHostToDevice));
            free(temp_grad);
        }
    }

    void Tensor::transpose() {
        const dim3 block(TILE_SIZE, TILE_SIZE);
        const dim3 grid((_shape[0] + block.x - 1) / block.x, (_shape[1] + block.y - 1) / block.y);
        value_type* temp;
        CHECK(hipMalloc(&temp, _size * sizeof(value_type)));
        krnl::Transpose(grid, block, _data, temp, _shape[0], _shape[1]);
        CHECK(hipDeviceSynchronize());
        CHECK(hipFree(_data));
        _data = temp;
        if (_requires_grad) {
            value_type* tempGrad;
            CHECK(hipMalloc(&tempGrad, _size * sizeof(value_type)));
            krnl::Transpose(grid, block, _grad, tempGrad, _shape[0], _shape[1]);
            CHECK(hipDeviceSynchronize());
            CHECK(hipFree(_grad));
            _grad = tempGrad;
        }
        std::swap(_shape[0], _shape[1]);
    }

    void Tensor::setData(const shape_type& position, const value_type value) const {
        if (position[0] >= _shape[0] || position[1] >= _shape[1]) {
            throw std::invalid_argument("Invalid position");
        }
        auto* data = static_cast<value_type*>(malloc(_size * sizeof(value_type)));
        CHECK(hipMemcpy(data, _data, _size * sizeof(value_type), hipMemcpyDeviceToHost));
        data[position[0] * _shape[1] + position[1]] = value;
        CHECK(hipMemcpy(_data, data, _size * sizeof(value_type), hipMemcpyHostToDevice));
        free(data);
    }

    Tensor::value_type* Tensor::data() const noexcept {
        return _data;
    }

    Tensor::value_type* Tensor::grad() const {
        if (!_requires_grad) {
            throw std::runtime_error("Tensor does not require gradients");
        }
        return _grad;
    }

    std::ostream& Tensor::printGrad(std::ostream& os) const {
        auto* data = static_cast<value_type*>(malloc(_size * sizeof(value_type)));
        CHECK(hipMemcpy(data, _grad, _size * sizeof(value_type), hipMemcpyDeviceToHost));
        std::ostream_iterator<value_type> output_iterator(os, " ");
        for (int i = 0; i < _shape[0]; ++i) {
            const auto it = data + i * _shape[1];
            const auto it_end = it + _shape[1];
            os << "[";
            std::copy(it, it_end, output_iterator);
            os << "]";
            os << std::endl;
        }
        free(data);
        return os;
    }

    Tensor Tensor::operator-() const {
        Tensor result(_shape, _requires_grad);
        const dim3 block(256);
        const dim3 grid((_size + block.x - 1) / block.x);
        krnl::Negation(grid, block, result._data, _data, _size);
        CHECK(hipDeviceSynchronize());
        return result;
    }

    void Tensor::recip() const {
        value_type* data;
        CHECK(hipMalloc(&data, _size * sizeof(value_type)));
        const dim3 block(256);
        const dim3 grid((_size + block.x - 1) / block.x);
        krnl::Recip(grid, block, data, _data, _size);
        CHECK(hipMemcpy(_data, data, _size * sizeof(value_type), hipMemcpyDeviceToDevice));
        hipFree(data);
    }

    Tensor::value_type Tensor::sum() const {
        const dim3 block(256);
        const dim3 grid((_size + block.x - 1) / block.x);
        value_type* dData;
        auto* hData = new value_type[grid.x];
        CHECK(hipMalloc(&dData, grid.x * sizeof(value_type)));
        krnl::Summation(grid, block, block.x / WARP_SIZE * sizeof(float), dData, _data, _size);
        CHECK(hipMemcpy(hData, dData, grid.x * sizeof(value_type), hipMemcpyDeviceToHost));
        value_type result = 0;
        for (auto i = 0; i < grid.x; ++i) {
            result += hData[i];
        }
        delete[] hData;
        CHECK(hipFree(dData));
        return result;
    }

    Tensor::value_type Tensor::expSum() const {
        const dim3 block(256);
        const dim3 grid((_size + block.x - 1) / block.x);
        value_type* dData;
        auto* hData = new value_type[grid.x];
        CHECK(hipMalloc(&dData, grid.x * sizeof(value_type)));
        krnl::SummationExp(grid, block, block.x / WARP_SIZE * sizeof(float), dData, _data, _size);
        CHECK(hipMemcpy(hData, dData, grid.x * sizeof(value_type), hipMemcpyDeviceToHost));
        value_type result = 0;
        for (auto i = 0; i < grid.x; ++i) {
            result += hData[i];
        }
        delete[] hData;
        CHECK(hipFree(dData));
        return result;
    }
}

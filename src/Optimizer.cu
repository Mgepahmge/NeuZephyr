//
// Created by Administrator on 24-11-20.
//

#include "NeuZephyr/Optimizer.cuh"

namespace NeuZephyr::Optimizers {
    SGD::SGD(const Tensor::value_type learning_rate) {
        this->learning_rate = learning_rate;
    }

    void SGD::step(Node* input) {
        dim3 block(256);
        dim3 grid((input->output->size() + block.x - 1) / block.x);
        SGD_kernel<<<grid, block>>>(input->output->data(), input->output->grad(), learning_rate, input->output->size());
    }

    Momentum::Momentum(Tensor::value_type learning_rate, Tensor::value_type beta) {
        this->learning_rate = learning_rate;
        this->beta = beta;
    }

    void Momentum::step(Node* input) {
        if (velocity.find(input) == velocity.end()) {
            Tensor v(input->output->shape(), false);
            v.fill(0);
            velocity[input] = v;
        }
        float* temp;
        hipMalloc(&temp, input->output->size() * sizeof(float));
        dim3 block(256);
        dim3 grid((input->output->size() + block.x - 1) / block.x);
        Momentum_kernel<<<grid, block>>>(temp, input->output->grad(), velocity[input].data(), beta,
                                         input->output->size());
        hipMemcpy(velocity[input].data(), temp, input->output->size() * sizeof(float), hipMemcpyDeviceToDevice);
        SGD_kernel<<<grid, block>>>(input->output->data(), velocity[input].data(), learning_rate,
                                    input->output->size());
        hipFree(temp);
    }

    AdaGrad::AdaGrad(Tensor::value_type learning_rate) {
        this->learning_rate = learning_rate;
    }

    void AdaGrad::step(Node* input) {
        if (G.find(input) == G.end()) {
            Tensor g(input->output->shape(), false);
            g.fill(0);
            G[input] = g;
        }
        dim3 block(256);
        dim3 grid((input->output->size() + block.x - 1) / block.x);
        AdaGrad_kernel<<<grid, block>>>(input->output->data(),  G[input].data(), input->output->grad(),learning_rate, epsilon, input->output->size());
    }
} // Optimizers
